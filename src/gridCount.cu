#include "hip/hip_runtime.h"
// **************************
// *                        *
// *  Sudoku 3x4 Cuda Code  *
// *                        *
// **************************

// This file is edited in Visual Studio by fooling it into thinking it's a cpp file,
// then copied to the Jetson and renamed from cpp to cu. These definitions aid that
// fooling, and are not seen by the Cuda toolchain, because JETSON is defined.
#ifndef JETSON
#define JETSON
#define __global__

struct dim3
{
  uint32_t x, y, z;
  dim3(uint32_t x = 1, uint32_t y = 1, uint32_t z = 1)
    : x(x), y(y), z(z)
  {}
}
threadIdx, blockIdx, blockDim, gridDim;

#endif

#include <stdint.h>
#include <cudapp.h>
#include <vector>
#include <array>
#include "sudokuda.h"

namespace GpuHost
{
  constexpr uint32_t DoubleBoxCount = ColCompatibleCount * ColCompatibleCount;

  dim3 threads(32);
  dim3 blocks(88);

  struct Box711
  {
    uint16_t box7;
    uint16_t box11;
    Box711() = default;
    Box711(uint16_t box7, uint16_t box11) : box7(box7), box11(box11) {}
  };

  // Box711 box711[ceil(groupSize/Coalesce)][ColCompatibleCount][Coalesce];
  // get(groupIndex, b3) = box711[groupIndex >> CoalesceBits][b3/Coalesce][groupIndex & CoalesceMaskLo]
  // thread(groupIndex, b3) = groupIndex & 31;
  // assume blockDim.x is power of 2
  constexpr uint32_t CoalesceBits = 2;
  constexpr uint32_t Coalesce = 1 << CoalesceBits;
  constexpr uint32_t CoalesceMaskLo = Coalesce - 1;
  constexpr uint32_t CoalesceMaskHi = ~CoalesceMaskLo;

  // ****************
  // *              *
  // *  Big Tables  *
  // *              *
  // ****************
  //
  // Huge tables created by CPU code and copied to device memory here. Also some device
  // and host arrays allocated here to avoid alloc/free operations in the counting loops.
  // All device and host memory collected here so it can be constructed and destroyed
  // easily.
  struct BigTables
  {
    // Device copy of BangGang::gangCache_, holding band counts at this stage
    CudaDeviceMemory<int32_t[ColCodeCount][ColCodeAligned]> gangCache{ 9 };

    // Device copy of GridCounter::codeCompatTable_
    const uint16_t(*codeCompatTableHost)[ColCompatibleCount][2];
    CudaDeviceMemory<uint16_t[ColCompatibleCount][2]> codeCompatTableDev{ ColCodeCount };

    // Device copy of GridCounter::gcPackets_
    const GridCountPacket* gcPacketsHost;
    CudaDeviceMemory<GridCountPacket> gcPacketsDev{ DoubleBoxCount };

    // list of box3 codes for current group
    CudaDeviceMemory<uint16_t> box3;

    // Device box7 and box11 codes
    CudaDeviceMemory<Box711> box711;

    // Partial counts, effectively uint64_t counts[blocks][box2GroupSize]
    CudaDeviceMemory<uint64_t> counts;

    int groupSize() const { return box3.numElements(); }

    std::vector<CudaStream> streams;
  }
  *bigTables = nullptr;

  void sudokudaEnd()
  {
    delete bigTables;
  }

  // **************************
  // *                        *
  // *  Print GPU Properties  *
  // *                        *
  // **************************

  void printDeviceProperties()
  {
    int deviceId = hipGetDevice(&deviceId);
    hipDeviceProp_t dp;
    checkErr(hipGetDeviceProperties(&dp, deviceId));

    printf("Clock rate %d kHz\n", dp.clockRate);
    printf("L2 cache size %d\n", dp.l2CacheSize);
    printf("Max blocks per multiprocessor %d\n", dp.maxBlocksPerMultiProcessor);
    printf("Max grid size %d.%d.%d\n", dp.maxGridSize[0], dp.maxGridSize[1], dp.maxGridSize[2]);
    printf("Max block dimension %d.%d.%d\n", dp.maxThreadsDim[0], dp.maxThreadsDim[1], dp.maxThreadsDim[2]);
    printf("Max threads per block %d\n", dp.maxThreadsPerBlock);
    printf("Max threads per multiprocessor %d\n", dp.maxThreadsPerMultiProcessor);
    printf("Multiprocessor count %d\n", dp.multiProcessorCount);
    printf("Reserved shared memory per block %d bytes\n", dp.reservedSharedMemPerBlock);
    printf("Shared memory per block %d bytes\n", dp.sharedMemPerBlock);
    printf("Shared memory per multiprocessor %d bytes\n", dp.sharedMemPerMultiprocessor);
    printf("Total global memory on device %lu bytes\n", dp.totalGlobalMem);
    printf("Warp size in threads %d \n", dp.warpSize);
  }

  // ************************************
  // *                                  *
  // *  Setup Functions Called by Host  *
  // *                                  *
  // ************************************

  void gpuGrid(uint32_t blocksX, uint32_t threadsX, uint32_t threadsY)
  {
    blocks.x = blocksX;
    threads.x = threadsX;
    threads.y = threadsY;
  }

  void gpuInit(const CacheLevel cache[],
               const uint16_t codeCompatTable[][ColCompatibleCount][2])
  {
    if (!bigTables)
      bigTables = new BigTables();
    bigTables->gangCache.copyTo((int32_t(*)[ColCodeCount][ColCodeAligned])cache);
    bigTables->codeCompatTableHost = codeCompatTable;
    bigTables->codeCompatTableDev.copyTo(codeCompatTable);
  }

  void gpuSetup(const GridCountPacket* gcPackets)
  {
    bigTables->gcPacketsHost = gcPackets;
    bigTables->gcPacketsDev.copyTo(gcPackets);
  }

  __global__
    void clearCounts(uint64_t* counts, int groupSize)
  {
    uint64_t* p = counts + blockIdx.x * groupSize;
    for (int groupIndex = threadIdx.x; groupIndex < groupSize; groupIndex += blockDim.x)
      p[groupIndex] = 0;
  }

  void gpuGroup(int box2GroupSize, const uint16_t* box3List)
  {
    bigTables->box3.alloc(box2GroupSize);
    bigTables->box3.copyTo(box3List);

    int32_t box711Size = ColCompatibleCount * ((box2GroupSize + CoalesceMaskLo) & CoalesceMaskHi);
    bigTables->box711.alloc(box711Size);
    bigTables->box711.clear();

    bigTables->counts.alloc(box2GroupSize * blocks.x);
    clearCounts<<<blocks, threads>>>(bigTables->counts.mem(), box2GroupSize);
  }

  void gpuAddGroup(uint64_t* groupCounts, int groupStride)
  {
    std::vector<uint64_t> counts(bigTables->counts.numElements());
    bigTables->counts.copyFrom(counts.data());

    int groupSize = bigTables->groupSize();

    for (int groupIndex = 0; groupIndex < groupSize; ++groupIndex)
    {
      uint64_t count = 0;
      for (int block = 0; block < blocks.x; ++block)
        count += counts[block * groupSize + groupIndex];
      groupCounts[groupIndex * groupStride] += count;
    }
  }

  // ***********************
  // *                     *
  // *  Box7/Box11 Kernel  *
  // *                     *
  // ***********************
  //
  // gridDim.x must be a multiple of Coalesce
  __global__
  void box711(int groupSize, const uint16_t(*codeCompatTable)[ColCompatibleCount][2],
              const uint16_t* doubleRename0, const uint16_t* doubleRename1,
              const uint16_t* box3List, Box711* box711)
  {
    box711 += blockIdx.x & CoalesceMaskLo;

    for (int groupIndex = blockIdx.x; groupIndex < groupSize; groupIndex += gridDim.x)
    {
      int box3 = box3List[groupIndex];
      const uint16_t(*compat)[2] = codeCompatTable[box3];

      Box711* box711Line = box711 + ColCompatibleCount * (CoalesceMaskHi & groupIndex);

      for (int b3 = threadIdx.x; b3 < ColCompatibleCount; b3 += blockDim.x)
      {
        int b7  = compat[b3][0];
        int b11 = compat[b3][1];
        Box711& bx = box711Line[b3 * Coalesce];
        bx.box7 = doubleRename0[b7];
        bx.box11 = doubleRename1[b11];
      }
    }
  }

  // **************************
  // *                        *
  // *  Main Counting Kernel  *
  // *                        *
  // **************************

  __global__
  void groupCount(const uint16_t(*codeCompatTable)[2],
                  const uint16_t* doubleRename0, const uint16_t* doubleRename1,
                  const int32_t (*cache0)[ColCodeAligned], const int32_t (*cache1)[ColCodeAligned],
                  const Box711* box711, int groupSize, int multiplier, uint64_t* counts)
  {
    uint64_t* p = counts + blockIdx.x * groupSize;
    box711 += threadIdx.x & CoalesceMaskLo;

    for (int b2 = blockIdx.x; b2 < ColCompatibleCount; b2 += gridDim.x)
    {
      int box6 = codeCompatTable[b2][0];
      int box10 = codeCompatTable[b2][1];

      box6  = doubleRename0[box6];
      box10 = doubleRename1[box10];

      const int32_t* band1CacheLine = cache0[box6 ];
      const int32_t* band2CacheLine = cache1[box10];

      for (int groupIndex = threadIdx.x; groupIndex < groupSize; groupIndex += blockDim.x)
      {
        const Box711* box711Line = box711 + (groupIndex & CoalesceMaskHi) * ColCompatibleCount;
        uint64_t count = 0;
        for (int b3 = 0; b3 < Coalesce * ColCompatibleCount; b3 += Coalesce)
          count += (uint64_t)band1CacheLine[box711Line[b3].box7] * band2CacheLine[box711Line[b3].box11];

        p[groupIndex] += count * multiplier;
      }
    }
  }

  // *****************************
  // *                           *
  // *  Main Counting Host Code  *
  // *                           *
  // *****************************

  void gpuMainCount(int box01, int box2)
  {
    const GridCountPacket& gcp0 = bigTables->gcPacketsHost[box01];

    int box01Other = gcp0.otherIndex;
    const GridCountPacket& gcp1 = bigTables->gcPacketsHost[box01Other];

    int groupSize = bigTables->groupSize();

    Box711* box711List = bigTables->box711.mem();

    const uint16_t* doubleRename0 = bigTables->gcPacketsDev[box01].doubleRename;
    const uint16_t* doubleRename1 = bigTables->gcPacketsDev[box01Other].doubleRename;

    box711<<<64, 64>>>(groupSize, bigTables->codeCompatTableDev.mem(),
                       doubleRename0, doubleRename1, bigTables->box3.mem(),
                       box711List);
    checkLaunch();

    const uint16_t(*codeCompatTable)[2] = bigTables->codeCompatTableDev[box2];

    const int32_t (*cache0)[ColCodeAligned] = bigTables->gangCache[gcp0.cacheLevel];
    const int32_t (*cache1)[ColCodeAligned] = bigTables->gangCache[gcp1.cacheLevel];

    int multiplier = bigTables->gcPacketsHost[box01].multiplier;

    uint64_t* counts = bigTables->counts.mem();
    //safeSync();

    groupCount<<<blocks, threads>>>(codeCompatTable, doubleRename0, doubleRename1,
                                    cache0, cache1, box711List,
                                    groupSize, multiplier, counts);
    checkLaunch();
    //safeSync();
  }
}